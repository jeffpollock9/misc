#include "hip/hip_runtime.h"
#include "add_cuda.hpp"

#include <cmath>

__global__ void
add_cuda_kernel(const int n, const float* x, const float* y, float* z)
{
    const int index  = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride)
    {
        z[i] = x[i] + y[i];
    }
}

void add_cuda(const int n, const float* x, const float* y, float* z)
{
    const int size = n * sizeof(float);

    float* x_d;
    float* yz_d;

    hipMalloc((void**)&x_d, size);
    hipMalloc((void**)&yz_d, size);

    hipMemcpy(x_d, x, size, hipMemcpyHostToDevice);
    hipMemcpy(yz_d, y, size, hipMemcpyHostToDevice);

    const int block_size = 256;
    const int n_blocks   = std::ceil(n / static_cast<float>(block_size));

    add_cuda_kernel<<<n_blocks, block_size>>>(n, x_d, yz_d, yz_d);

    hipDeviceSynchronize();

    hipMemcpy(z, yz_d, size, hipMemcpyDeviceToHost);

    hipFree(x_d);
    hipFree(yz_d);
}